#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include <sys/time.h>
#include<time.h>

#define min(a,b) (a<b?a:b)
#define TILE_WIDTH 4

typedef struct GpuLstmStore {
   //for partial matrix vector results of gates for current and next time steps
   double *i,*f,*g,*o;
   double *c,*h; //present time step computations
   int evenOrOdd; //0 = even, 1 = Odd;

   double *onChipI; //on-chip storage for fixing part of Ri,Rf,Rg,Ro matrices data
   double *onChipF; //on-chip storage for fixing part of Ri,Rf,Rg,Ro matrices data
   double *onChipG; //on-chip storage for fixing part of Ri,Rf,Rg,Ro matrices data
   double *onChipO; //on-chip storage for fixing part of Ri,Rf,Rg,Ro matrices data
   int squareDim; // dimensions beyond which data is stored on-chip
}GpuLstmStore;

typedef struct GpuLstmLayer{
   int inputSize;
   int numHiddenUnits;
   double *Wi,*Wf,*Wg,*Wo;
   double *Ri,*Rf,*Rg,*Ro;
   double *bi,*bf,*bg,*bo;
   double *hiddenState;
   double *cellState;

   double *xinp;
   double *xi,*xf,*xg,*xo;

   GpuLstmStore store; //storage for temporary variables
}GpuLstmLayer;

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


double* readMatrixFromFile( FILE *fp, int R, int C)
{
    double *buff = NULL;

   if (fp != NULL)
   {
      buff = (double*)malloc(sizeof(double) * R * C);

      for (int i=0;i<R;i++)
      {
         for (int j=0;j<C;j++)
         {
            if(fscanf(fp, "%lf", &buff[i*C+j]) != 1)
               exit(1);
         }
      }
   }

   return buff;
}


void initializeMatrix(double *mat, int R, int C)
{
    srand(time(0));

    for (int i=0;i<R;i++)
    {
        for (int j=0;j<C;j++)
        {
            mat[i*C+j] = (rand()%10000) * 0.0001;
        }
    }
}

__global__  void  gpu_matrixVectorMultWX(double *dmatA, double *dvecB, double *dvecC, int matrow,int matcol)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*blockDim.x*tidy;


    if(tindex<matrow)
    {
      int i;
      int m=tindex*matcol;
      double sum = 0.0;
    
     for(i=0;i<matcol;i++)
     {
         sum += dmatA[m+i] * dvecB[i];
     }

     dvecC[tindex] = sum;
  
    }

      // __syncthreads();
}



/////////////////////////////////////////////////
   ////BLOCKWISE MAIN CODE ////

__device__ void getChunkOfArray(double *temp, double *store, int index, int rCeil, int numHiddenUnits)
{
   int rind = index * TILE_WIDTH;

   for (int i=0;i<TILE_WIDTH;i++)
   {
       temp[i] = store[rind + i];
   }
}

__device__ void getBlockOfArray2D(double *temp, double *store, int rowNo, int colNo, int rCeil, int numHiddenUnits)
{
   int rind = rowNo * TILE_WIDTH;
   int cind = colNo * TILE_WIDTH;
 
   for (int i=0;i<TILE_WIDTH;i++)
   {
       for (int j=0;j<TILE_WIDTH;j++)
       {
           temp[i * TILE_WIDTH + j] = store[ ((rind+i) * numHiddenUnits) + (cind+j)];
       }
   }
}


__device__ void getBlockChunkMultiply(double *temp, double *block, double *chunk, int rowNo, int colNo, int rCeil, int numHiddenUnits)
{
    int rind = rowNo * TILE_WIDTH;
 
    for (int i=0;i<TILE_WIDTH;i++)
    {
        double sum = 0.0;
        for (int j=0;j<TILE_WIDTH;j++)
        {
            sum += (block[i * TILE_WIDTH + j] * chunk[j]);
        }
        
        temp[rind+i] += sum;
    }
}


__device__ double sigmoid_elementwise(double val)
{
    return 1/(1+exp(-val));
}

__device__ void sigmoid_blockwise(double *temp, double *x, double *bias, int rowNo, int rCeil, int numHiddenUnits)
{
    int rind = rowNo * TILE_WIDTH;

    for (int i=0;i<TILE_WIDTH;i++)
    {
        temp[rind+i] = sigmoid_elementwise(temp[rind+i] + x[rind+i] + bias[rind + i]);
    }
}

__device__ void tanh_blockwise(double *temp, double *x, double *bias, int rowNo, int rCeil, int numHiddenUnits)
{
    int rind = rowNo * TILE_WIDTH;

    for (int i=0;i<TILE_WIDTH;i++)
    {
        temp[rind+i] = tanh(temp[rind+i] + x[rind+i] + bias[rind+i]);
    }
}

__device__ void cellAndHiddenVal_blockwise(double *hval, double *cval, double *devCpr,
                                           double *iPres, double *fPres, double *gPres, double *oPres,
                                           int rowNo, int rCeil, int numHiddenUnits)
{
    int rind = rowNo * TILE_WIDTH;
 
    for (int i=0;i<TILE_WIDTH;i++)
    {
        cval[rind+i] = (fPres[rind+i] * devCpr[rind+i]) + (iPres[rind+i] * gPres[rind+i]);
        hval[rind+i] = oPres[rind+i] * tanh(cval[rind+i]);
    }
}

__device__ void assignNextGateValue(double *store, double *temp, int rowNo, int rCeil, int numHiddenUnits)
{
    int rind = rowNo * TILE_WIDTH;

    for (int i=0;i<TILE_WIDTH;i++)
    {
        store[rind+i] = temp[rind+i];
    }
}

__device__ void assignZeroGateValue(double *store, int rowNo, int rCeil, int numHiddenUnits)
{
    int rind = rowNo * TILE_WIDTH;

    for (int i=0;i<TILE_WIDTH;i++)
    {
        store[rind+i] = 0.0;
    }
}

//// Maximum size of hiddenunits for which shared memory will not overflow for the case of double type ////
/// for variant5, SMEM_SIZE = 768
/// for varinat6 elementwise, SMEM_SIZE = 767
/// for varinat4 blockwise
/// if TILE_WIDTH = 4, SMEM_SIZE = 764
/// if TILE_WIDTH = 8, SMEM_SIZE = 760
/// if TILE_WIDTH = 16, SMEM_SIZE = 752
/// if TILE_WIDTH = 32, SMEM_SIZE = 736
/// if TILE_WIDTH = 64, SMEM_SIZE = 704


#define SMEM_SIZE 764

__global__ void gateValueComputation_Cond0(double *devHpr, double *devCpr,
                                           double *devRi, double *devRf, double *devRg, double *devRo,
                                           double *devBi, double *devBf, double *devBg, double *devBo,
                                           double *xi, double *xf, double *xg, double *xo,
                                           double *ival, double *fval, double *gval, double *oval, double *cval, double *hval,
                                           int rCeil, int numHiddenUnits)
{
    
   __shared__ double iPres[SMEM_SIZE], fPres[SMEM_SIZE], gPres[SMEM_SIZE], oPres[SMEM_SIZE];
   __shared__ double iNext[SMEM_SIZE], fNext[SMEM_SIZE], gNext[SMEM_SIZE], oNext[SMEM_SIZE];
   __shared__ double tmpHpr[TILE_WIDTH];
   __shared__ double tmphval[TILE_WIDTH];

   int index = blockDim.x * blockIdx.x + threadIdx.x;

   if (index == 0)
   {
       for (int i=0;i<numHiddenUnits;i++)
       {
           iPres[i] = ival[i];
           fPres[i] = fval[i];
           gPres[i] = gval[i];
           oPres[i] = oval[i];

           iNext[i] = 0.0;
           fNext[i] = 0.0;
           gNext[i] = 0.0;
           oNext[i] = 0.0;
       }
   }
 
   __syncthreads();
 
   for (int i=0;i<rCeil;i++)
   {
       if (index == i)
       {
           // double tmpHpr[TILE_WIDTH];
           
           getChunkOfArray(tmpHpr, devHpr, i, rCeil, numHiddenUnits);
        
           double tRi[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRi, devRi, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(iPres, tRi, tmpHpr, index, i, rCeil, numHiddenUnits);

           double tRf[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRf, devRf, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(fPres, tRf, tmpHpr, index, i, rCeil, numHiddenUnits);
        
           double tRg[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRg, devRg, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(gPres, tRg, tmpHpr, index, i, rCeil, numHiddenUnits);
        
           double tRo[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRo, devRo, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(oPres, tRo, tmpHpr, index, i, rCeil, numHiddenUnits);
        
           sigmoid_blockwise(iPres, xi, devBi, index, rCeil, numHiddenUnits);
           sigmoid_blockwise(fPres, xf, devBf, index, rCeil, numHiddenUnits);
           tanh_blockwise(gPres, xg, devBg, index, rCeil, numHiddenUnits);
           sigmoid_blockwise(oPres, xo, devBo, index, rCeil, numHiddenUnits);

           cellAndHiddenVal_blockwise(hval, cval, devCpr, iPres, fPres, gPres, oPres, index, rCeil, numHiddenUnits);

           // double tmphval[TILE_WIDTH];
        
           getChunkOfArray(tmphval, hval, i, rCeil, numHiddenUnits);
           
           getBlockChunkMultiply(iNext, tRi, tmphval, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(fNext, tRf, tmphval, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(gNext, tRg, tmphval, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(oNext, tRo, tmphval, index, i, rCeil, numHiddenUnits);
        
           assignNextGateValue(ival, iNext, index, rCeil, numHiddenUnits);
           assignNextGateValue(fval, fNext, index, rCeil, numHiddenUnits);
           assignNextGateValue(gval, gNext, index, rCeil, numHiddenUnits);
           assignNextGateValue(oval, oNext, index, rCeil, numHiddenUnits);
        
       }

       __syncthreads();
    
       if (index > i)
       {
           // double tmpHpr[TILE_WIDTH];
           // double tmphval[TILE_WIDTH];

           // getChunkOfArray(tmpHpr, devHpr, i, rCeil, numHiddenUnits);
           // getChunkOfArray(tmphval, hval, i, rCeil, numHiddenUnits);

           double tRi[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRi, devRi, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(iPres, tRi, tmpHpr, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(iNext, tRi, tmphval, index, i, rCeil, numHiddenUnits);

           double tRf[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRf, devRf, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(fPres, tRf, tmpHpr, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(fNext, tRf, tmphval, index, i, rCeil, numHiddenUnits);

           double tRg[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRg, devRg, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(gPres, tRg, tmpHpr, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(gNext, tRg, tmphval, index, i, rCeil, numHiddenUnits);

           double tRo[TILE_WIDTH * TILE_WIDTH];
           getBlockOfArray2D(tRo, devRo, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(oPres, tRo, tmpHpr, index, i, rCeil, numHiddenUnits);
           getBlockChunkMultiply(oNext, tRo, tmphval, index, i, rCeil, numHiddenUnits);

       }

       __syncthreads();
   }

}  


__global__ void outputElementComputation_Cond1( double *devCpr, double *xi, double *xf, double *xg, double *xo,
                                               double *devBi, double *devBf, double *devBg, double *devBo,
                                               double *ival, double *fval, double *gval, double *oval, double *cval, double *hval,
                                               int rindex, int rCeil, int numHiddenUnits)
{
    
    sigmoid_blockwise(ival, xi, devBi, rindex, rCeil, numHiddenUnits);
    sigmoid_blockwise(fval, xf, devBf, rindex, rCeil, numHiddenUnits);
    tanh_blockwise(gval, xg, devBg, rindex, rCeil, numHiddenUnits);
    sigmoid_blockwise(oval, xo, devBo, rindex, rCeil, numHiddenUnits);

    cellAndHiddenVal_blockwise(hval, cval, devCpr, ival, fval, gval, oval, rindex, rCeil, numHiddenUnits);

    if (rindex == rCeil-1)
    {
        assignZeroGateValue(ival, rindex, rCeil, numHiddenUnits);
        assignZeroGateValue(fval, rindex, rCeil, numHiddenUnits);
        assignZeroGateValue(gval, rindex, rCeil, numHiddenUnits);
        assignZeroGateValue(oval, rindex, rCeil, numHiddenUnits);
     
    }
 
}


__global__ void gateValueComputation_Cond1( double *devHpr, double *devCpr,
                                          double *devRi, double *devRf, double *devRg, double *devRo,
                                          double *devBi, double *devBf, double *devBg, double *devBo,
                                          double *xi, double *xf, double *xg, double *xo,
                                          double *ival, double *fval, double *gval, double *oval, double *cval, double *hval,
                                          int rCeil, int numHiddenUnits)
{


    __shared__ double iPres[SMEM_SIZE], fPres[SMEM_SIZE], gPres[SMEM_SIZE], oPres[SMEM_SIZE];
    __shared__ double iNext[SMEM_SIZE], fNext[SMEM_SIZE], gNext[SMEM_SIZE], oNext[SMEM_SIZE];
    __shared__ double tmpHpr[TILE_WIDTH];
    __shared__ double tmphval[TILE_WIDTH];

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index == 0)
    {
        for (int i=0;i<numHiddenUnits;i++)
        {
            iPres[i] = ival[i];
            fPres[i] = fval[i];
            gPres[i] = gval[i];
            oPres[i] = oval[i];

            iNext[i] = 0.0;
            fNext[i] = 0.0;
            gNext[i] = 0.0;
            oNext[i] = 0.0;
        }
     
        getChunkOfArray(tmpHpr, devHpr, rCeil-1, rCeil, numHiddenUnits);
        getChunkOfArray(tmphval, hval, rCeil-1, rCeil, numHiddenUnits);
    }

    __syncthreads();

    for (int i=rCeil-1;i>=1;i--)
    {
        if (index < i)
        {
            //double tmpHpr[TILE_WIDTH];
            //double tmphval[TILE_WIDTH];
         
            // getChunkOfArray(tmpHpr, devHpr, i, rCeil, numHiddenUnits);
            // getChunkOfArray(tmphval, hval, i, rCeil, numHiddenUnits);

            double tRi[TILE_WIDTH * TILE_WIDTH];
            getBlockOfArray2D(tRi, devRi, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(iPres, tRi, tmpHpr, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(iNext, tRi, tmphval, index, i, rCeil, numHiddenUnits);

            double tRf[TILE_WIDTH * TILE_WIDTH];
            getBlockOfArray2D(tRf, devRf, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(fPres, tRf, tmpHpr, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(fNext, tRf, tmphval, index, i, rCeil, numHiddenUnits);
         
            double tRg[TILE_WIDTH * TILE_WIDTH];
            getBlockOfArray2D(tRg, devRg, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(gPres, tRg, tmpHpr, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(gNext, tRg, tmphval, index, i, rCeil, numHiddenUnits);

            double tRo[TILE_WIDTH * TILE_WIDTH];
            getBlockOfArray2D(tRo, devRo, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(oPres, tRo, tmpHpr, index, i, rCeil, numHiddenUnits);
            getBlockChunkMultiply(oNext, tRo, tmphval, index, i, rCeil, numHiddenUnits);
         
        }

        __syncthreads();

        if (index == i-1)
        {
            sigmoid_blockwise(iPres, xi, devBi, index, rCeil, numHiddenUnits);
            sigmoid_blockwise(fPres, xf, devBf, index, rCeil, numHiddenUnits);
            tanh_blockwise(gPres, xg, devBg, index, rCeil, numHiddenUnits);
            sigmoid_blockwise(oPres, xo, devBo, index, rCeil, numHiddenUnits);

            cellAndHiddenVal_blockwise(hval, cval, devCpr, iPres, fPres, gPres, oPres, index, rCeil, numHiddenUnits);

            assignNextGateValue(ival, iNext, index, rCeil, numHiddenUnits);
            assignNextGateValue(fval, fNext, index, rCeil, numHiddenUnits);
            assignNextGateValue(gval, gNext, index, rCeil, numHiddenUnits);
            assignNextGateValue(oval, oNext, index, rCeil, numHiddenUnits);

            getChunkOfArray(tmpHpr, devHpr, index, rCeil, numHiddenUnits);
            getChunkOfArray(tmphval, hval, index, rCeil, numHiddenUnits);

        }

        __syncthreads();
    }

}   


void LSTMForwardStepBlockReuse(double *x, GpuLstmLayer *gpulstmlayer)
{
   int inputSize = gpulstmlayer->inputSize;
   int numHiddenUnits = gpulstmlayer->numHiddenUnits;
   int condition = gpulstmlayer->store.evenOrOdd;

   if (numHiddenUnits%TILE_WIDTH != 0 || TILE_WIDTH > numHiddenUnits || SMEM_SIZE < numHiddenUnits)
   {
       printf(" wrong input of inputsize and number of hidden units \n");
       return;
   }

   double *devX;
   double *devWi, *devWf, *devWg, *devWo;
   double *devRi, *devRf, *devRg, *devRo;
   double *devBi, *devBf, *devBg, *devBo;
   double *xi, *xf, *xg, *xo;
   double *hval, *cval, *ival, *fval, *gval, *oval;
   double *devHpr, *devCpr;
 
   /// doing gpu memory allocation and copying cpu data into gpu memory
   hipMemcpy(gpulstmlayer->xinp, x, sizeof(double) * inputSize, hipMemcpyHostToDevice);
   devX = gpulstmlayer->xinp;
 
   ///
   devWi = gpulstmlayer->Wi;
   devWf = gpulstmlayer->Wf;
   devWg = gpulstmlayer->Wg;
   devWo = gpulstmlayer->Wo;
 
   ///
   devRi = gpulstmlayer->Ri;
   devRf = gpulstmlayer->Rf;
   devRg = gpulstmlayer->Rg;
   devRo = gpulstmlayer->Ro;

   devBi = gpulstmlayer->bi;
   devBf = gpulstmlayer->bf;
   devBg = gpulstmlayer->bg;
   devBo = gpulstmlayer->bo;
 

   ///
   xi = gpulstmlayer->xi;
   xf = gpulstmlayer->xf;
   xg = gpulstmlayer->xg;
   xo = gpulstmlayer->xo;
    
   ///
   hval = gpulstmlayer->store.h;
   cval = gpulstmlayer->store.c;
   ival = gpulstmlayer->store.i;
   fval = gpulstmlayer->store.f;
   gval = gpulstmlayer->store.g;
   oval = gpulstmlayer->store.o;

   /// 
   devHpr = gpulstmlayer->hiddenState;
   devCpr = gpulstmlayer->cellState;
 

    /// doing block and grid setting for multiplication functions
    int blockSize = 16;
    int maxSize = blockSize * blockSize;
    int blocksPerGrid = numHiddenUnits/maxSize + 1;
    dim3 dimBlock(blockSize, blockSize);

    if (numHiddenUnits%maxSize == 0)
    {
        blocksPerGrid--;
    }
 
    dim3 dimGrid(1, blocksPerGrid);
 
    //// computing xi, xf, xg, xo, by matrix vector multiplication of W and x
   gpu_matrixVectorMultWX<<< dimGrid , dimBlock >>>(devWi, devX, xi, numHiddenUnits, inputSize );
   gpu_matrixVectorMultWX<<< dimGrid , dimBlock >>>(devWf, devX, xf, numHiddenUnits, inputSize );
   gpu_matrixVectorMultWX<<< dimGrid , dimBlock >>>(devWg, devX, xg, numHiddenUnits, inputSize );
   gpu_matrixVectorMultWX<<< dimGrid , dimBlock >>>(devWo, devX, xo, numHiddenUnits, inputSize );


   if (condition == 0)
   {
        int rCeil = (numHiddenUnits + TILE_WIDTH - 1)/TILE_WIDTH;
    
        int threadInBlock = rCeil;
        int linBlockPerGrid = 1;
    
        gateValueComputation_Cond0<<< linBlockPerGrid , threadInBlock >>>(devHpr, devCpr, devRi, devRf, devRg, devRo, devBi, devBf, devBg, devBo,
                                                                              xi, xf, xg, xo, ival, fval, gval, oval, cval, hval, rCeil, numHiddenUnits);
    
   }
   else if (condition == 1)
   { 
       int rCeil = (numHiddenUnits + TILE_WIDTH - 1)/TILE_WIDTH;

       outputElementComputation_Cond1<<< 1, 1 >>>( devCpr, xi, xf, xg, xo, devBi, devBf, devBg, devBo, ival, fval, gval, oval, cval, hval, rCeil-1, rCeil, numHiddenUnits);

       int threadInBlock = rCeil-1;
       int linBlockPerGrid = 1;

       if (threadInBlock != 0)
       {
           gateValueComputation_Cond1<<< linBlockPerGrid , threadInBlock >>>(devHpr, devCpr, devRi, devRf, devRg, devRo, devBi, devBf, devBg, devBo,
                                                                              xi, xf, xg, xo, ival, fval, gval, oval, cval, hval, rCeil, numHiddenUnits);   
       }
   }
   else
   {
       printf(" Wrong input \n");
       return;
   }
    

   //// changing value of condition for next iteration 
   gpulstmlayer->store.evenOrOdd = 1 - condition;

   /// copy operations 
   hipMemcpy(devHpr, hval, sizeof(double) * numHiddenUnits, hipMemcpyDeviceToDevice);
   hipMemcpy(devCpr, cval, sizeof(double) * numHiddenUnits, hipMemcpyDeviceToDevice);

   /// re-initialization operations
   //cudaMemset(hval, 0, sizeof(double) * numHiddenUnits);
   //cudaMemset(cval, 0, sizeof(double) * numHiddenUnits);

   /// free gpu variables
  
}


///////////////////////////////////////////////

void createGpuLSTMLayer(GpuLstmLayer *gpulstmlayer,int numHiddenUnits,int inputSize,
                                            double *W,double *R,double *bias)
{
    ///
    hipMalloc(&(gpulstmlayer->Wi), sizeof(double)*numHiddenUnits * inputSize);
    hipMemcpy(gpulstmlayer->Wi, W, sizeof(double) * numHiddenUnits * inputSize, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->Wf), sizeof(double)*numHiddenUnits * inputSize);
    hipMemcpy(gpulstmlayer->Wf, W + numHiddenUnits*inputSize, sizeof(double) * numHiddenUnits * inputSize, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->Wg), sizeof(double)*numHiddenUnits * inputSize);
    hipMemcpy(gpulstmlayer->Wg, W + 2 * numHiddenUnits*inputSize, sizeof(double) * numHiddenUnits * inputSize, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->Wo), sizeof(double)*numHiddenUnits * inputSize);
    hipMemcpy(gpulstmlayer->Wo, W + 3 * numHiddenUnits*inputSize, sizeof(double) * numHiddenUnits * inputSize, hipMemcpyHostToDevice);
 
    ///
    hipMalloc(&(gpulstmlayer->Ri), sizeof(double)*numHiddenUnits * numHiddenUnits);
    hipMemcpy(gpulstmlayer->Ri, R, sizeof(double) * numHiddenUnits * numHiddenUnits, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->Rf), sizeof(double)*numHiddenUnits * numHiddenUnits);
    hipMemcpy(gpulstmlayer->Rf, R + numHiddenUnits * numHiddenUnits, sizeof(double) * numHiddenUnits * numHiddenUnits, hipMemcpyHostToDevice);
 
    hipMalloc(&(gpulstmlayer->Rg), sizeof(double)*numHiddenUnits * numHiddenUnits);
    hipMemcpy(gpulstmlayer->Rg, R + 2 * numHiddenUnits * numHiddenUnits, sizeof(double) * numHiddenUnits * numHiddenUnits, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->Ro), sizeof(double)*numHiddenUnits * numHiddenUnits);
    hipMemcpy(gpulstmlayer->Ro, R + 3 * numHiddenUnits * numHiddenUnits, sizeof(double) * numHiddenUnits * numHiddenUnits, hipMemcpyHostToDevice);

    ////
    hipMalloc(&(gpulstmlayer->bi), sizeof(double) * numHiddenUnits);
    hipMemcpy(gpulstmlayer->bi, bias, sizeof(double) * numHiddenUnits, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->bf), sizeof(double) * numHiddenUnits);
    hipMemcpy(gpulstmlayer->bf, bias + numHiddenUnits, sizeof(double) * numHiddenUnits, hipMemcpyHostToDevice);
 
    hipMalloc(&(gpulstmlayer->bg), sizeof(double) * numHiddenUnits);
    hipMemcpy(gpulstmlayer->bg, bias + 2 * numHiddenUnits, sizeof(double) * numHiddenUnits, hipMemcpyHostToDevice);

    hipMalloc(&(gpulstmlayer->bo), sizeof(double) * numHiddenUnits);
    hipMemcpy(gpulstmlayer->bo, bias + 3 * numHiddenUnits, sizeof(double) * numHiddenUnits, hipMemcpyHostToDevice);

    //initialize the cell state and hidden units
    hipMalloc(&(gpulstmlayer->cellState), sizeof(double) * numHiddenUnits);
    hipMalloc(&(gpulstmlayer->hiddenState), sizeof(double) * numHiddenUnits);
 
    hipMemset(gpulstmlayer->cellState, 0, sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->hiddenState, 0, sizeof(double) * numHiddenUnits);
    
    gpulstmlayer->inputSize = inputSize;
    gpulstmlayer->numHiddenUnits = numHiddenUnits;

    //memory for gates, even and odd 
    hipMalloc(&(gpulstmlayer->store.i), sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->store.i, 0, sizeof(double) * numHiddenUnits);
 
    hipMalloc(&(gpulstmlayer->store.f), sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->store.f, 0, sizeof(double) * numHiddenUnits);
 
    hipMalloc(&(gpulstmlayer->store.g), sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->store.g, 0, sizeof(double) * numHiddenUnits);

    hipMalloc(&(gpulstmlayer->store.o), sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->store.o, 0, sizeof(double) * numHiddenUnits);
 
    hipMalloc(&(gpulstmlayer->store.c), sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->store.c, 0, sizeof(double) * numHiddenUnits);
 
    hipMalloc(&(gpulstmlayer->store.h), sizeof(double) * numHiddenUnits);
    hipMemset(gpulstmlayer->store.h, 0, sizeof(double) * numHiddenUnits);
 
    hipMalloc(&(gpulstmlayer->xinp), sizeof(double) * inputSize);
    
    hipMalloc(&(gpulstmlayer->xi), sizeof(double) * numHiddenUnits);
    hipMalloc(&(gpulstmlayer->xf), sizeof(double) * numHiddenUnits);
    hipMalloc(&(gpulstmlayer->xg), sizeof(double) * numHiddenUnits);
    hipMalloc(&(gpulstmlayer->xo), sizeof(double) * numHiddenUnits);
 

    gpulstmlayer->store.evenOrOdd = 0; //start with even, lower diagonal matrix.
}



void freeGpuLSTMLayer(GpuLstmLayer *gpulstmlayer)
{
    hipFree(gpulstmlayer->Wi);
    hipFree(gpulstmlayer->Wf);
    hipFree(gpulstmlayer->Wg);
    hipFree(gpulstmlayer->Wo);

    hipFree(gpulstmlayer->Ri);
    hipFree(gpulstmlayer->Rf);
    hipFree(gpulstmlayer->Rg);
    hipFree(gpulstmlayer->Ro);

    hipFree(gpulstmlayer->bi);
    hipFree(gpulstmlayer->bf);
    hipFree(gpulstmlayer->bg);
    hipFree(gpulstmlayer->bo);

    hipFree(gpulstmlayer->store.i);
    hipFree(gpulstmlayer->store.f);
    hipFree(gpulstmlayer->store.g);
    hipFree(gpulstmlayer->store.o);

    hipFree(gpulstmlayer->store.c);
    hipFree(gpulstmlayer->store.h);
 
    hipFree(gpulstmlayer->xinp);
 
    hipFree(gpulstmlayer->xi);
    hipFree(gpulstmlayer->xf);
    hipFree(gpulstmlayer->xg);
    hipFree(gpulstmlayer->xo);

    // free(lstmlayer->Wi); //these were allocated outside createLSTMLayer
    // free(lstmlayer->Ri);//these were allocated outside createLSTMLayer
    // free(lstmlayer->bi);
    hipFree(gpulstmlayer->hiddenState);
    hipFree(gpulstmlayer->cellState);
}


void writeMatrixIntoFile( FILE *fp, double *mat, int R, int C)
{
   for (int i=0;i<R;i++)
   {
      for (int j=0;j<C;j++)
      {
         fprintf(fp, "%lf ", mat[i*C+j] );
      }
      fprintf(fp,"\n");
   }
}

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d", stat);
  return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime) {
  printf("%s%lf seconds\n", str, endtime - starttime);
}


int main()
{
   printf(" Variant4 (doing computation left to right (cond0) and right to left (cond1) and better use of shared memory.) of Method 3 :-  GPU blockwise implementation  method :- \n");
   int inputSize, numHiddenUnits, timeSteps;

   // char Fx[40], FinWts[40], FrecWts[40], Fbias[40] ;
   // char Fy[40];

   // printf("Enter the parameters of LSTM layer :- \n");

   // printf(" Enter the value of input size : ");
   // scanf("%d", &inputSize);
   // printf(" Enter the value of number of hidden units : ");
   // scanf("%d", &numHiddenUnits);
   // printf(" Enter the value of number of time Steps : ");
   // scanf("%d", &timeSteps);

   // printf("\n");
   
   // printf(" file name containing input vector : ");
   // scanf("%s", Fx);
   // printf(" file name containing input matrix weights :");
   // scanf("%s", FinWts);
   // printf(" file name containing hidden matrix weights : ");
   // scanf("%s", FrecWts);
   // printf(" file name containing bias vector : ");
   // scanf("%s", Fbias);
   // printf(" file name containing output vector : ");
   // scanf("%s", Fy);

   inputSize = 764;
   numHiddenUnits = 764;
   timeSteps = 4;
   int tileSize = TILE_WIDTH;
   int smemSize = SMEM_SIZE;
 
   printf(" inputSize = %d , numHiddenUnits = %d , timeSteps = %d , tileSize = %d , smemSize = %d \n", inputSize, numHiddenUnits, timeSteps, tileSize, smemSize);

   // strcpy( Fx, "Fx_8_2.txt");
   // strcpy( FinWts, "FinWts_8_8.txt");
   // strcpy( FrecWts, "Frec_8_8.txt");
   // strcpy( Fbias, "Fbias_8_1.txt");
   // strcpy( Fy, "Fy_gpu_conv_8_2.txt");

   // FILE *fpX = fopen(Fx, "r");
   // FILE *fpInWts = fopen(FinWts, "r");
   // FILE *fpRecWts = fopen(FrecWts, "r");
   // FILE *fpBias = fopen(Fbias, "r");
   // FILE *fpY = fopen(Fy, "w");

   // if ((fpX == NULL) || (fpInWts == NULL) || ( fpRecWts == NULL) || (fpBias == NULL) )
   // {
      // printf(" file opening error ");
      // return -1;
   // }

   double *X, *W, *R, *bias;
   double *Y;

    // X = readMatrixFromFile( fpX, timeSteps, inputSize);
    // W = readMatrixFromFile( fpInWts, 4 * numHiddenUnits, inputSize);
    // R = readMatrixFromFile( fpRecWts, 4 * numHiddenUnits, numHiddenUnits);
    // bias = readMatrixFromFile( fpBias, 4 * numHiddenUnits, 1);

    // fclose( fpX);
    // fclose( fpInWts);
    // fclose( fpRecWts);
    // fclose( fpBias);

    W = (double*)malloc(sizeof(double) * 4 * numHiddenUnits * inputSize);
    R = (double*)malloc(sizeof(double) * 4 * numHiddenUnits * numHiddenUnits);
    bias = (double*)malloc(sizeof(double) * 4 * numHiddenUnits * 1);
    X = (double*)malloc(sizeof(double) * inputSize * timeSteps);
    Y = (double*)malloc(sizeof(double) * numHiddenUnits * timeSteps);

    initializeMatrix(W, 4*numHiddenUnits, inputSize);
    initializeMatrix(R, 4*numHiddenUnits, numHiddenUnits);
    initializeMatrix(bias, 4*numHiddenUnits, 1);
    initializeMatrix(X, inputSize, timeSteps);
    // initializeMatrix(Y, numHiddenUnits, timeSteps);
    memset(Y, 0, sizeof(numHiddenUnits * timeSteps));

    // Y = (double*)malloc(sizeof(double) * numHiddenUnits * timeSteps);
    // memset(Y, 0, sizeof(double) * numHiddenUnits * timeSteps);

    GpuLstmLayer gpulstmlayer;
    createGpuLSTMLayer(&gpulstmlayer,numHiddenUnits,inputSize,W,R,bias);

    double total_time = 0.0;
 
    //double startTime , endTime;
    struct timeval t1, t2;
 
    for (int i=0;i<timeSteps;i++)
    {
      // startTime = rtclock();
     
      gettimeofday(&t1, 0);
      LSTMForwardStepBlockReuse(&X[i*inputSize],&gpulstmlayer);
      hipMemcpy(&Y[i*numHiddenUnits],gpulstmlayer.hiddenState,sizeof(double)*numHiddenUnits, hipMemcpyDeviceToHost);
      
      //endTime = rtclock();
      gettimeofday(&t2, 0);
     
      double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
      
      // total_time += endTime - startTime;
     
      total_time += time;
      printf(" time for this step is %lf milliseconds \n ", time);
      printf("time after %d time step is %lf milliseconds \n", i, total_time );
    }
    printf("\n");
 
    


    printf("total time taken by gpu blockwise program (variant4 - doing computation left to right (cond0) and right to left (cond1) and better use of shared memory) is %lf milliseconds for input size of %d , numHiddenUnits of %d , timeSteps of %d and tileSize of %d \n", total_time, inputSize, numHiddenUnits, timeSteps, tileSize );
    // writeMatrixIntoFile(fpY,  Y, timeSteps, inputSize);
    // fclose(fpY);

    freeGpuLSTMLayer( &gpulstmlayer);
    free(W);
    free(R);
    free(bias);
    free(X);
    free(Y);
}